#include "hip/hip_runtime.h"
#include "material.h"
#include "quad.h"
#include "scenes-wk2.h"
#include "sphere.h"
#include "texture.h"
#include "utils.h"

__global__ void create_earth(bvh_node *d_bvh_nodes, hitable **d_list, camera *d_camera,
                             unsigned char *d_pixel_data, int width, int height, int channels,
                             int list_size, int nx, int ny)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i > 0) || (j > 0))
        return;

    auto earth_texture = new rtapp::image_texture(d_pixel_data, width, height, channels);
    d_list[0] = new sphere(vec3(0, 0, 0), 2.0, new lambertian(earth_texture));

    // create bvh_nodes
    bvh_node::prefill_nodes(d_bvh_nodes, d_list, list_size);

    *d_camera = camera();
    d_camera->lookfrom = vec3(-9, -2, -10);
    d_camera->lookat = vec3(0, 0, 0);
    d_camera->vup = vec3(0, 1, 0);
    d_camera->vfov = 20.0;
    d_camera->image_width = nx;
    d_camera->image_height = ny;
    d_camera->defocus_angle = 0.0;
    d_camera->initialize();
}

void earth(bvh_node *&h_bvh_nodes, bvh_node *&d_bvh_nodes, hitable **&d_list, camera *&d_camera, int &list_size, int &tree_size, int nx, int ny)
{
    auto earth_texture = rtapp::image_texture("assets/earthmap.jpg");

    // copy texture to device
    unsigned char *d_pixel_data;
    checkCudaErrors(hipMalloc((void **)&d_pixel_data, earth_texture.pixel_data_size));
    checkCudaErrors(hipMemcpy(d_pixel_data, earth_texture.pixel_data, earth_texture.pixel_data_size, hipMemcpyHostToDevice));

    list_size = 1;
    checkCudaErrors(hipMalloc((void **)&d_list, list_size * sizeof(hitable *)));

    tree_size = 2 * list_size;
    h_bvh_nodes = new bvh_node[tree_size]; // binary tree
    checkCudaErrors(hipMalloc((void **)&d_bvh_nodes, tree_size * sizeof(bvh_node)));

    create_earth<<<dim3(1, 1), dim3(1, 1)>>>(d_bvh_nodes, d_list, d_camera,
                                             d_pixel_data, earth_texture.width, earth_texture.height, earth_texture.channels,
                                             list_size, nx, ny);

    std::cout << "earth scene created" << std::endl;
}

__global__ void create_two_perlin_spheres(bvh_node *d_bvh_nodes, hitable **d_list, camera *d_camera,
                                          int list_size, int nx, int ny)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i > 0) || (j > 0))
        return;

    auto perlin_texture = new rtapp::noise_texture(4.0);
    d_list[0] = new sphere(vec3(0, -1000, 0), 1000, new lambertian(perlin_texture));
    d_list[1] = new sphere(vec3(0, 2, 0), 2, new lambertian(perlin_texture));

    // create bvh_nodes
    bvh_node::prefill_nodes(d_bvh_nodes, d_list, list_size);

    *d_camera = camera();
    d_camera->lookfrom = vec3(13, 2, 3);
    d_camera->lookat = vec3(0, 0, 0);
    d_camera->vup = vec3(0, 1, 0);
    d_camera->vfov = 20.0;
    d_camera->image_width = nx;
    d_camera->image_height = ny;
    d_camera->defocus_angle = 0.0;
    d_camera->initialize();
}

void two_perlin_spheres(bvh_node *&h_bvh_nodes, bvh_node *&d_bvh_nodes, hitable **&d_list, camera *&d_camera, int &list_size, int &tree_size, int nx, int ny)
{
    INIT_LIST_AND_TREE(2);

    create_two_perlin_spheres<<<dim3(1, 1), dim3(1, 1)>>>(d_bvh_nodes, d_list, d_camera,
                                                          list_size, nx, ny);
}

__global__ void create_quads(bvh_node *d_bvh_nodes, hitable **d_list, camera *d_camera,
                             int list_size, int nx, int ny)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i > 0) || (j > 0))
        return;

    // Materials
    auto left_red = new lambertian(vec3(1.0, 0.2, 0.2));
    auto back_green = new lambertian(vec3(0.2, 1.0, 0.2));
    auto right_blue = new lambertian(vec3(0.2, 0.2, 1.0));
    auto upper_orange = new lambertian(vec3(1.0, 0.5, 0.0));
    auto lower_teal = new lambertian(vec3(0.2, 0.8, 0.8));

    // Quads
    d_list[0] = new quad(vec3(-3, -2, 5), vec3(0, 0, -4), vec3(0, 4, 0), left_red);
    d_list[1] = new quad(vec3(-2, -2, 0), vec3(4, 0, 0), vec3(0, 4, 0), back_green);
    d_list[2] = new quad(vec3(3, -2, 1), vec3(0, 0, 4), vec3(0, 4, 0), right_blue);
    d_list[3] = new quad(vec3(-2, 3, 1), vec3(4, 0, 0), vec3(0, 0, 4), upper_orange);
    d_list[4] = new quad(vec3(-2, -3, 5), vec3(4, 0, 0), vec3(0, 0, -4), lower_teal);

    // create bvh_nodes
    bvh_node::prefill_nodes(d_bvh_nodes, d_list, list_size);

    *d_camera = camera();
    d_camera->lookfrom = vec3(0, 0, 9);
    d_camera->lookat = vec3(0, 0, 0);
    d_camera->vup = vec3(0, 1, 0);
    d_camera->vfov = 80.0;
    d_camera->image_width = nx;
    d_camera->image_height = ny;
    d_camera->defocus_angle = 0.0;
    d_camera->initialize();
}

void quads(bvh_node *&h_bvh_nodes, bvh_node *&d_bvh_nodes, hitable **&d_list, camera *&d_camera, int &list_size, int &tree_size, int nx, int ny)
{
    INIT_LIST_AND_TREE(5);

    create_quads<<<dim3(1, 1), dim3(1, 1)>>>(d_bvh_nodes, d_list, d_camera,
                                             list_size, nx, ny);
}
