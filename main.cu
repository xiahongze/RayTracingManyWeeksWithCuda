#include "hip/hip_runtime.h"
#include "cmd_parser.h"
#include "image_utils.h"
#include "render.h"
#include "scenes.h"
#include "utils.h"
#include "vec3.h"
#include <float.h>
#include <iostream>
#include <time.h>

int main(int argc, char **argv)
{
    auto cmd_opts = parse_command_line(argc, argv);

    int num_pixels = cmd_opts.image_width * cmd_opts.image_height;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // make our world of hitables & the camera
    hitable **d_list;
    int list_size, tree_size;

    // create two arrays of bvh_nodes on host and device
    bvh_node *h_bvh_nodes, *d_bvh_nodes;

    camera *d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera)));

    int scene = 0;

    switch (scene)
    {
    case 0:
        random_spheres(h_bvh_nodes, d_bvh_nodes, d_list, d_camera, list_size, tree_size,
                       cmd_opts.image_width, cmd_opts.image_height, cmd_opts.bounce, cmd_opts.bounce_pct, cmd_opts.checkered);
        break;
    default:
        exit(1);
    }

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // copy bvh_nodes from device to host
    checkCudaErrors(hipMemcpy(h_bvh_nodes, d_bvh_nodes, list_size * sizeof(bvh_node), hipMemcpyDeviceToHost));
    // build bvh tree on host
    int tree_height = bvh_node::build_tree(h_bvh_nodes, list_size);
    // copy bvh_nodes from host to device
    checkCudaErrors(hipMemcpy(d_bvh_nodes, h_bvh_nodes, tree_size * sizeof(bvh_node), hipMemcpyHostToDevice));

    clock_t start, stop;
    start = clock();

    // Render our buffer
    dim3 blocks(cmd_opts.image_width / cmd_opts.tx + (cmd_opts.image_width % cmd_opts.tx ? 1 : 0),
                cmd_opts.image_height / cmd_opts.ty + (cmd_opts.image_height % cmd_opts.ty ? 1 : 0));
    dim3 threads(cmd_opts.tx, cmd_opts.ty);
    render<<<blocks, threads>>>(fb, cmd_opts.image_width, cmd_opts.image_height, cmd_opts.samples_per_pixel, d_camera, d_bvh_nodes);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::clog << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image, allocated with hipMallocManaged can be directly accessed on host
    writeJPGImage(cmd_opts.output_file.c_str(), cmd_opts.image_width, cmd_opts.image_height, fb);

    // clean up
    free_objects<<<dim3(1), dim3(32)>>>(d_list, list_size);
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_bvh_nodes));
    checkCudaErrors(hipFree(fb));
    delete[] h_bvh_nodes;

    hipDeviceReset();
}
