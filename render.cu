#include "hip/hip_runtime.h"
#include "material.h"
#include "render.h"
#include "utils.h"
#include "vec3.h"
#include <hiprand/hiprand_kernel.h>

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 get_ray_color_pixel(const int max_depth, const ray &r, bvh_node *d_bvh_nodes, hitable_list **d_lights, vec3 &backgroound, hiprandState *local_rand_state)
{
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    vec3 final_color(0, 0, 0);
    for (int i = 0; i < max_depth; i++)
    {
        hit_record rec;
        if (!bvh_node::hit(d_bvh_nodes, cur_ray, interval(0.001f, FLT_MAX), rec, local_rand_state))
        {
            final_color += backgroound * cur_attenuation;
            break;
        }

        scatter_record srec;

        vec3 color_from_emission = rec.mat_ptr->emitted(r, rec, rec.u, rec.v, rec.p);

        if (!rec.mat_ptr->scatter(cur_ray, rec, srec, local_rand_state))
        {
            final_color += color_from_emission * cur_attenuation;
            break;
        }

        if (srec.skip_pdf)
        {
            cur_attenuation *= srec.attenuation;
            cur_ray = srec.skip_pdf_ray;
            continue;
        }

        // do sample
        pdf *p_cur;
        switch (srec.pdf_type_)
        {
        case pdf_type::COSINE:
            p_cur = &srec.cosine_pdf_;
            break;
        case pdf_type::HITABLE:
            p_cur = &srec.hitable_pdf_;
            break;
        case pdf_type::MIXTURE:
            p_cur = &srec.mixture_pdf_;
            break;
        default:
            p_cur = &srec.sphere_pdf_;
            break;
        }

        pdf *p;
        hitable_pdf pdf_light(*d_lights, rec.p);
        mixture_pdf p_mixed(p_cur, &pdf_light);

        if (*d_lights != nullptr && (*d_lights)->length() > 0) // if there is light
        {
            p = &p_mixed;
        }
        else
        {
            p = p_cur;
        }

        ray scattered = ray(rec.p, p->generate(local_rand_state), r.get_time());
        auto pdf_val = p->value(scattered.direction(), local_rand_state);

        float scattering_pdf = rec.mat_ptr->scattering_pdf(r, rec, scattered);

        cur_ray = scattered;
        if (pdf_val > 1e-6)
        {
            // avoid divide by zero
            cur_attenuation *= srec.attenuation * scattering_pdf / pdf_val;
        }
    }
    return final_color; // exceeded recursion
}

__global__ void render(vec3 *d_fb, int max_x, int max_y, int ns, int max_depth, int rand_seed, camera *d_camera, bvh_node *d_bvh_nodes, hitable_list **d_lights)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y))
        return;
    int pixel_index = j * max_x + i;

    int sqrt_spp = static_cast<int>(sqrt(ns));
    float recip_sqrt_spp = 1.0 / sqrt_spp;

    hiprandState local_rand_state;
    hiprand_init(rand_seed + pixel_index, 0, 0, &local_rand_state);

    vec3 col(0, 0, 0);
    for (int s_j = 0; s_j < sqrt_spp; ++s_j)
    {
        for (int s_i = 0; s_i < sqrt_spp; ++s_i)
        {
            ray r = d_camera->get_ray(i, j, s_i, s_j, recip_sqrt_spp, &local_rand_state);
            // can call vec3.clamp() here but not here because it help with debugging purpose
            col += get_ray_color_pixel(max_depth, r, d_bvh_nodes, d_lights, d_camera->background, &local_rand_state);
        }
    }
    col /= float(ns);
    col.to_gamma_space();
    d_fb[pixel_index] = col;
}